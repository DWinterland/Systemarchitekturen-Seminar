#include "hip/hip_runtime.h"
#include <stdlib.h>
#include <hip/hip_runtime_api.h>
#include "kernel.h"
#include <limits.h>
#include <math.h>


/**
 * The inner computational kernel (will be executed on the GPU).
 *
 * @param       lbDev   Left image (global memory)
 * @param       rbDev   Right image (global memory)
 * @param	zeilen	number of rows of the image
 * @param	spalten	number of columns of the image
 * @param       b       width of the correlation window
 * @param       h       height of the correlation window
 * @param       tauMax  Maximal dispersion
 * @param       useS    Use a threshold (Schwellwert)
 * @param       useF    Use weighted window (please ignore)
 * @param       s       threshold to apply (when useS==true)
 * @param       profileDev Dispersion profile in global memory (to be filled by the algorithm)
 * @param       validDev   Validation matrix in global memory (to be filled by the algorithm)
 */
__global__ void StereoKernel(char *lbDev,char *rbDev, int zeilen, int spalten,int tauMax, int *profileDev,
	      unsigned char *validDev, int b, int h, bool useS, bool useF, int s)
{
   // TODO: Implement the kernel!
   
   
		int xu = -b / 2;
		int xo = (b % 2 == 1) ? b / 2 : b / 2 - 1;
		int yu = -h / 2;
		int yo = (h % 2 == 1) ? h / 2 : h / 2 - 1;
		int optIndex;
		int optWert;
		int wert;
		bool val;

		int i = blockIdx.x * blockDim.x + threadIdx.x;
		int j = blockIdx.y * blockDim.y + threadIdx.y;
		if (i >= zeilen ||j >= spalten) return;
		//for (int i = 0; i < zeilen; i++) {
				//for (int j = 0; j < spalten; j++) {

				if ((i + yu < 0) | (i + yo >= zeilen) | (j + xu - tauMax < 0)
						| (j + xo + tauMax >= spalten)) {
					profileDev[i * spalten + j] = 0;
					validDev[i * spalten + j] = 0;
				} else {
					optWert = 0xFFFF;
					optIndex = 0;
					val = false;
					for (int tau = -tauMax; tau <= tauMax; tau++) {
						wert = 0;
						for (int k = xu; k <= xo; k++) {
							for (int l = yu; l <= yo; l++) {
								int left = lbDev[(i + l) * spalten + j + k] & 0xFF;
								int right = rbDev[(i + l) * spalten + j + k + tau] & 0xFF;  
								wert += abs(left-right);
							}
						}
						if (wert < optWert) {
							optWert = wert;
							optIndex = tau;
							val = true;
						} else if (wert == optWert) {
							val = false;
						}
					}
					profileDev[i * spalten + j] = optIndex;
					validDev[i * spalten + j] = val;
				}
			//}
		//}
}

/**
 * Perform the stereodiff algorithm in CUDA.
 *
 * @param       lb      Left image
 * @param       rb      Right image
 * @param	zeilen	number of rows of the image
 * @param	spalten	number of columns of the image
 * @param       b       width of the correlation window
 * @param       h       height of the correlation window
 * @param       tauMax  Maximal dispersion
 * @param       useS    Use a threshold (Schwellwert)
 * @param       useF    Use weighted window (please ignore)
 * @param       s       threshold to apply (when useS==true)
 * @param       profile Dispersion profile (to be filled by the algorithm)
 * @param       valid  Validation matrix (to be filled by the algorithm)
 */
void doCUDACalc(signed char *lb, signed char *rb, int zeilen, int spalten,int tauMax, int *profile,
 unsigned char *valid, int b, int h, bool useS, bool useF, int s)
{


	int num_devices, device;
	hipGetDeviceCount(&num_devices);
	if (num_devices > 1) {
		int max_multiprocessors = 0, max_device = 0;
		for (device = 0; device < num_devices; device++) {
			hipDeviceProp_t properties;
			hipGetDeviceProperties(&properties, device);
			if (max_multiprocessors < properties.multiProcessorCount) {
				max_multiprocessors = properties.multiProcessorCount;
				max_device = device;
			}
		}
		hipSetDevice(max_device);
	}

	char *lbDev; // left image in CUDA memory
	char *rbDev; // right image in CUDA memory
	int *profileDev; // profile matrix. to be filled by the algorithm
	unsigned char *validDev; // valid matrix. to be filled by the algorithm

	// allocate global memory for the GPU
	checkCudaErrors(hipMalloc((void**)&lbDev, zeilen*spalten*sizeof(char)));
	checkCudaErrors(hipMalloc((void**)&rbDev, zeilen*spalten*sizeof(char)));
	checkCudaErrors(hipMalloc((void**)&profileDev, zeilen*spalten*sizeof(int)));
	checkCudaErrors(hipMalloc((void**)&validDev, zeilen*spalten*sizeof(unsigned char)));

	printf("Copying data to CUDA memory\n");
	checkCudaErrors(hipMemcpy(lbDev, lb, spalten*zeilen, hipMemcpyHostToDevice));
	checkCudaErrors(hipMemcpy(rbDev, rb, spalten*zeilen, hipMemcpyHostToDevice));


	// TODO: call the kernel
	
	dim3 dimBlock(16,16);
	dim3 dimGrid((zeilen + dimBlock.x - 1) / dimBlock.x, (spalten + dimBlock.y - 1) / dimBlock.y);
 
	StereoKernel<<<dimGrid,dimBlock>>>(lbDev,rbDev, zeilen, spalten, tauMax, profileDev, validDev, b, h, useS, useF, s);

	printf("Copying results back from CUDA memory\n");
	checkCudaErrors(hipMemcpy(valid, validDev, spalten*zeilen, hipMemcpyDeviceToHost));
	checkCudaErrors(hipMemcpy(profile, profileDev, spalten*zeilen*sizeof(int), hipMemcpyDeviceToHost));

	// free memory
	checkCudaErrors(hipFree(lbDev));
	checkCudaErrors(hipFree(rbDev));
	checkCudaErrors(hipFree(profileDev));
	checkCudaErrors(hipFree(validDev));

        hipDeviceReset();

}


